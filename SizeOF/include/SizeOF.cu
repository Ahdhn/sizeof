#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "gtest/gtest.h"

struct MyStruct
{
    ~MyStruct() = default;
    uint32_t m;
};

struct MyStructV
{
    virtual ~MyStructV() = default;
    uint32_t m;
};

TEST(Test, SizeOF)
{
    std::cout << "sizeof(MyStruct)= " << sizeof(MyStruct) << std::endl;
    std::cout << "sizeof(MyStructV)= " << sizeof(MyStructV) << std::endl;
}

int main(int argc, char** argv)
{
    ::testing::InitGoogleTest(&argc, argv);

    return RUN_ALL_TESTS();
}
